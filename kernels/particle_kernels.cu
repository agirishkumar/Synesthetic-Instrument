#include "hip/hip_runtime.h"
// kernels/particle_kernels.cu
#include "particle_kernels.h"
#include <hip/hip_runtime.h>
#include "config.h"
#include <math.h>

__constant__ float3 GRAVITY = {0.0f, GRAVITY_STRENGTH, 0.0f};

// Helper functions for vector operations
__device__ float3 operator+(const float3 &a, const float3 &b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator*(const float3 &v, float s)
{
    return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ void operator+=(float3 &a, const float3 &b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}

__device__ void operator*=(float3 &v, float s)
{
    v.x *= s;
    v.y *= s;
    v.z *= s;
}

__device__
    float4
    hsv_to_rgb(float h, float s, float v)
{
    float c = v * s;
    float hh = fmodf(h, 360.0f) / 60.0f;
    float x = c * (1.0f - fabsf(fmodf(hh, 2.0f) - 1.0f));
    float m = v - c;

    float4 rgba = {0.0f, 0.0f, 0.0f, 1.0f};

    if (hh <= 1.0f)
    {
        rgba.x = c;
        rgba.y = x;
    }
    else if (hh <= 2.0f)
    {
        rgba.x = x;
        rgba.y = c;
    }
    else if (hh <= 3.0f)
    {
        rgba.y = c;
        rgba.z = x;
    }
    else if (hh <= 4.0f)
    {
        rgba.y = x;
        rgba.z = c;
    }
    else if (hh <= 5.0f)
    {
        rgba.x = x;
        rgba.z = c;
    }
    else
    {
        rgba.x = c;
        rgba.z = x;
    }

    rgba.x += m;
    rgba.y += m;
    rgba.z += m;

    return rgba;
}

__device__
    float3
    apply_audio_influence(float3 base_pos, const AudioParams *audio,
                          float frequency, EffectType effect)
{
    float3 influenced = base_pos;

    // Apply different effects based on frequency ranges
    switch (effect)
    {
    case EFFECT_SPIRAL:
        influenced.x *= (1.0f + audio->mid_magnitude * 0.5f);
        influenced.y *= (1.0f + audio->treble_magnitude * 0.3f);
        influenced.z *= (1.0f + audio->bass_magnitude * 0.7f);
        break;

    case EFFECT_FOUNTAIN:
        influenced.y *= (1.0f + audio->bass_magnitude * 1.2f);
        influenced.x *= (1.0f + audio->mid_magnitude * 0.4f);
        influenced.z *= (1.0f + audio->mid_magnitude * 0.4f);
        break;

    case EFFECT_WAVE:
        influenced.y *= (1.0f + audio->bass_magnitude * 0.8f);
        influenced.x *= (1.0f + audio->treble_magnitude * 0.3f);
        break;

    case EFFECT_EXPLOSION:
        float intensity = (audio->bass_magnitude + audio->mid_magnitude) * 0.5f;
        influenced.x *= (1.0f + intensity * 0.6f);
        influenced.y *= (1.0f + intensity * 0.6f);
        influenced.z *= (1.0f + intensity * 0.6f);
        break;
    }

    return influenced;
}

__device__ void generate_spiral_effect(Particle *particle, int note, float velocity, float angle_offset, int pattern_idx)
{
    float angle = note * (2.0f * M_PI / 127.0f) + angle_offset;
    float radius = 0.5f + velocity * 0.3f;
    float height_offset = sinf(angle * 3.0f) * 0.2f;

    particle->position = make_float3(
        cosf(angle) * radius,
        height_offset,
        sinf(angle) * radius);

    particle->velocity = make_float3(
        cosf(angle) * velocity * 1.5f,
        velocity * 2.0f + cosf(angle) * 0.5f,
        sinf(angle) * velocity * 1.5f);
}

__device__ void generate_fountain_effect(Particle *particle, int note, float velocity, float angle_offset, int pattern_idx)
{
    float angle = note * (2.0f * M_PI / 127.0f) + angle_offset;
    float spread = pattern_idx * 0.2f;

    particle->position = make_float3(
        cosf(angle) * spread * 0.3f,
        -1.0f,
        sinf(angle) * spread * 0.3f);

    particle->velocity = make_float3(
        cosf(angle) * velocity * 2.0f,
        velocity * 4.0f,
        sinf(angle) * velocity * 2.0f);
}

__device__ void generate_wave_effect(Particle *particle, int note, float velocity, float angle_offset, int pattern_idx)
{
    float base_x = ((float)pattern_idx / PARTICLES_PER_NOTE - 0.5f) * 2.0f;
    float wave_height = sinf(base_x * M_PI * 2.0f + angle_offset);

    particle->position = make_float3(
        base_x,
        wave_height * 0.3f - 0.5f,
        0.0f);

    particle->velocity = make_float3(
        velocity * cosf(wave_height) * 0.5f,
        velocity * 2.0f,
        velocity * sinf(base_x * M_PI) * 0.5f);
}

__device__ void generate_explosion_effect(Particle *particle, int note, float velocity, float angle_offset, int pattern_idx)
{
    float phi = pattern_idx * (M_PI / PARTICLES_PER_NOTE);
    float theta = angle_offset + note * (2.0f * M_PI / 127.0f);

    particle->position = make_float3(0.0f, 0.0f, 0.0f);

    particle->velocity = make_float3(
        velocity * sinf(phi) * cosf(theta) * 3.0f,
        velocity * cosf(phi) * 3.0f,
        velocity * sinf(phi) * sinf(theta) * 3.0f);
}

__global__ void update_particles(Particle *particles, int count, float delta_time,
                                 const AudioParams *audio_params)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count || !particles[idx].active)
        return;

    if (particles[idx].lifetime > 0.0f)
    {
        // Apply custom forces based on effect type
        switch (particles[idx].effect_type)
        {
        case EFFECT_SPIRAL:
            particles[idx].velocity.y += sinf(particles[idx].lifetime * 5.0f +
                                              audio_params->mid_magnitude * 10.0f) *
                                         0.01f;
            break;

        case EFFECT_FOUNTAIN:
            float3 gravity = {0.0f, GRAVITY_STRENGTH * (1.0f + audio_params->bass_magnitude), 0.0f};
            particles[idx].velocity += gravity * delta_time;
            break;

        case EFFECT_WAVE:
            particles[idx].velocity.x *= 0.98f;
            particles[idx].velocity.z = cosf(particles[idx].lifetime * 10.0f +
                                             audio_params->treble_magnitude * 5.0f) *
                                        0.1f;
            break;

        case EFFECT_EXPLOSION:
            particles[idx].velocity *= (0.99f - audio_params->bass_magnitude * 0.02f);
            break;
        }

        // Apply audio-reactive modifications to velocity
        particles[idx].velocity = apply_audio_influence(
            particles[idx].velocity,
            audio_params,
            particles[idx].frequency,
            particles[idx].effect_type);

        // Update position
        float3 delta_pos = particles[idx].velocity * delta_time;
        particles[idx].position += delta_pos;

        // Add audio-reactive turbulence
        float turbulence = sinf(particles[idx].lifetime * 4.0f + particles[idx].position.x) *
                           (0.02f + audio_params->mid_magnitude * 0.03f);
        particles[idx].position.x += turbulence;
        particles[idx].position.z += turbulence;

        // Update lifetime and color
        particles[idx].lifetime -= delta_time;
        float life_ratio = particles[idx].lifetime / particles[idx].initial_lifetime;

        // Modify alpha based on audio
        float audio_alpha = (audio_params->bass_magnitude * 0.3f +
                             audio_params->mid_magnitude * 0.4f +
                             audio_params->treble_magnitude * 0.3f);
        particles[idx].color.w = (life_ratio * 0.6f + 0.2f) * (1.0f + audio_alpha * 0.5f);

        // Modify size based on audio
        float size_mult = 1.0f + audio_params->global_intensity * 0.5f;
        particles[idx].size = particles[idx].initial_size *
                              (life_ratio * 0.8f + 0.2f) * size_mult;

        // Deactivate if needed
        if (particles[idx].lifetime <= 0.0f ||
            fabsf(particles[idx].position.x) > 2.0f ||
            fabsf(particles[idx].position.y) > 2.0f ||
            fabsf(particles[idx].position.z) > 2.0f)
        {
            particles[idx].active = false;
        }
    }
}

__global__ void handle_note_event(Particle *particles, int note, float velocity,
                                  int start_idx, int count, int channel,
                                  const AudioParams *audio_params)
{
    int local_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (local_idx >= count)
        return;

    int idx = start_idx + local_idx;
    if (velocity > 0.01f)
    {
        float angle_offset = local_idx * (2.0f * M_PI / count);

        // Choose effect type based on MIDI channel and audio intensity
        EffectType effect = static_cast<EffectType>(channel % NUM_EFFECTS);
        particles[idx].effect_type = effect;

        // Generate initial position and velocity based on effect
        switch (effect)
        {
        case EFFECT_SPIRAL:
            generate_spiral_effect(&particles[idx], note, velocity, angle_offset, local_idx);
            break;
        case EFFECT_FOUNTAIN:
            generate_fountain_effect(&particles[idx], note, velocity, angle_offset, local_idx);
            break;
        case EFFECT_WAVE:
            generate_wave_effect(&particles[idx], note, velocity, angle_offset, local_idx);
            break;
        case EFFECT_EXPLOSION:
            generate_explosion_effect(&particles[idx], note, velocity, angle_offset, local_idx);
            break;
        }

        // Store original velocity for audio influence
        particles[idx].base_speed = sqrtf(
            particles[idx].velocity.x * particles[idx].velocity.x +
            particles[idx].velocity.y * particles[idx].velocity.y +
            particles[idx].velocity.z * particles[idx].velocity.z);

        // Set frequency for audio reactivity
        particles[idx].frequency = 440.0f * powf(2.0f, (note - 69) / 12.0f);

        // Set color based on note and audio
        float hue = fmodf(note * 2.8f + effect * 90.0f +
                              audio_params->global_intensity * 180.0f,
                          360.0f);
        particles[idx].color = hsv_to_rgb(hue,
                                          0.8f + velocity * 0.2f,
                                          0.8f + audio_params->global_intensity * 0.2f);

        // Set other properties with audio influence
        float lifetime_mult = 1.0f + audio_params->global_intensity * 0.5f;
        particles[idx].initial_lifetime = PARTICLE_LIFETIME *
                                          (0.5f + velocity * 0.5f) * lifetime_mult;
        particles[idx].lifetime = particles[idx].initial_lifetime;

        float size_mult = 1.0f + audio_params->global_intensity * 0.3f;
        particles[idx].initial_size = (MIN_PARTICLE_SIZE +
                                       (MAX_PARTICLE_SIZE - MIN_PARTICLE_SIZE) * velocity) *
                                      size_mult;
        particles[idx].size = particles[idx].initial_size;

        particles[idx].active = true;
    }
}

__global__ void reset_particles(Particle *particles, int count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    particles[idx].active = false;
    particles[idx].lifetime = 0.0f;
    particles[idx].color.w = 0.0f;
}